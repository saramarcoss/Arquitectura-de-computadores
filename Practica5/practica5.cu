#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <assert.h> 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#define N 10000000 // tamaño del vector
#define MAX_ERR 1e-6

global void vector_add(float *out, float *a, float *b, int n) { // realiza la suma de vectores en el dispositivo con el kernel vector_add 
int index = threadIdx.x; int stride = blockDim.x;
for(int i = index; i < n; i += stride){ 
    out[i] = a[i] + b[i];
    }
}

int main(){
float *a, *b, *out; // declaracion de punteros para los vectores a, b y out en el host
float *d_a, *d_b, *d_out; // declaracion de punteros para los vectores a, b y out en el dispositivo

// Allocate host memory
a = (float*)malloc(sizeof(float) * N); // reserva memoria en el host para el vector a. 
b = (float*)malloc(sizeof(float) * N); // reserva memoria en el host para el vector b
out = (float*)malloc(sizeof(float) * N); // reserva memoria en el host para el vector out

// Initialize host arrays
for(int i = 0; i < N; i++){
     a[i] = 1.0f; 
     b[i] = 2.0f; 
}

// Allocate device memory 
hipMalloc((void**)&d_a, sizeof(float) * N);  // reserva memoria en el dispositivo para el vector a. cuda
hipMalloc((void**)&d_b, sizeof(float) * N);  // reserva memoria en el dispositivo para el vector b
hipMalloc((void**)&d_out, sizeof(float) * N); // reserva memoria en el dispositivo para el vector out

// Transfer data from host to device memory
hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);  // copia el vector a al dispositivo. hipMemcpyHostToDevice: copia de host a dispositivo
hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice); // copia el vector b al dispositivo


blockDim.x = 256;// con blockDim.x = 256, se ejecutará el kernel vector_add con 256 hilos por bloque
gridDim.x = (2*N + blockDim.x - 1) / blockDim.x; 

// Executing kernel 
vector_add<<< gridDim.x, blockDim.x >>>(d_out, d_a, d_b, N); // ejecuta el kernel vector_add en el dispositivo con el número de bloques y el número de hilos por bloque

// Transfer data back to host memory 
hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost); // copia el vector out del dispositivo al host  hipMemcpyDeviceToHost: copia de dispositivo a host

// Verification
for(int i = 0; i < N; i++){ // comprueba que el resultado es correcto 
assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR); // fabs: devuelve el valor absoluto de un número. assert: comprueba que la condición es verdadera. Si no lo es, se produce un error y se muestra un mensaje de error
} // out - a - b = 0
printf("PASSED\n");

// Deallocate device memory 
hipFree(d_a);  // libera la memoria del vector a en el dispositivo
hipFree(d_b);  // libera la memoria del vector b en el dispositivo
hipFree(d_out); // libera la memoria del vector out en el dispositivo

// Deallocate host memory 
free(a); // libera la memoria del vector a en el host
free(b);  // libera la memoria del vector b en el host
free(out); // libera la memoria del vector out en el host
}